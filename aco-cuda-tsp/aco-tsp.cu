#include <assert.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <math.h>
#include <sstream>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// ACO constants
#define ANTS 1024
#define ALPHA 2
#define BETA 10
#define RHO 0.5
#define Q 50
#define MAX_ITERATIONS 10

// Instance constants
#define NODES 105
#define DIST 10000
#define PHERO_INITIAL (1.0 / NODES)
#define TOTAL_DIST (DIST * NODES)

// Base structure for ants information
struct ant {
  int curNode, nextNode, pathIndex;
  int tabu[NODES];
  int solution[NODES];
  float solutionLen;
};

struct nodeTSP {
  float x, y;
};

using namespace std;

// Host variables
float *heuristic;
double *phero;
struct ant antColony[ANTS];
float bestSol[ANTS];
float globalBest = TOTAL_DIST;
hiprandState state[ANTS];
const size_t heuristic_size = sizeof(float) * size_t(NODES * NODES);
const size_t phero_size = sizeof(double) * size_t(NODES * NODES);

// Device variables
float *heuristic_d;
double *phero_d;
struct ant *antColony_d;
float *bestSol_d;
hiprandState *state_d;
int BLOCKS, THREADS;

// Function headers
__global__ void initializeAnts(struct ant *antColony_d, hiprandState *state_d,
                               float *bestSol_d);
__global__ void setuCurandStates(hiprandState *stated_d, unsigned long t,
                                 float *bestSol_d);
__global__ void restartAnts(struct ant *antColony_d, hiprandState *state_d,
                            float *bestSol_d);
__global__ void constructSolution(struct ant *antColony_d, hiprandState *state_d, float *heuristic_d, double *phero_d);
__global__ void atomicUpdate(struct ant *antColony_d, double *phero_d);
__device__ double probFunctionProduct(int from, int to, double *phero_d,float *heuristic_d);
__device__ int NextNode(struct ant *antColony_d, int pos, float *heuristic_d,double *phero_d, hiprandState *state_d);

float euclideanDistance(float x1, float x2, float y1, float y2) {
  float xd = x1 - x2;
  float yd = y1 - y2;
  return (float)(sqrt(xd * xd + yd * yd));
}

void constructTSP(string graph, nodeTSP *nodes) {
  ifstream infile(("instances/" + graph + ".tsp").c_str());
  string line;
  bool euclidean = true;
  int node;
  float x, y;
  bool reading_nodes = false;

  // check all file lines
  while (getline(infile, line)) {
    istringstream iss(line);
    string word;
    if (!reading_nodes) {
      iss >> word;
      if (word.compare("EDGE_WEIGHT_TYPE") == 0) {
        iss >> word >> word;
         cout << "edge type: " << word << endl;
        euclidean = !word.compare("EUC_2D");
      } else if (word.compare("NODE_COORD_SECTION") == 0) {
        reading_nodes = true;
      }
    } else if (iss >> node >> x >> y) {
      nodes[node - 1].x = x;
      nodes[node - 1].y = y;
    }
  }
  infile.close();
  // Calculate distances between cities (edge weights)
  for (int from = 0; from < NODES; from++) {
    for (int to = from + 1; to < NODES; to++) {
      float edge_weight;
      if (euclidean) {
        edge_weight = euclideanDistance(nodes[from].x, nodes[to].x,
                                        nodes[from].y, nodes[to].y);
      }

      if (edge_weight == 0) {
        edge_weight = 1.0;
      }
      heuristic[from +to * NODES] = edge_weight;
      heuristic[to + from * NODES] = edge_weight;
      phero[from + to * NODES] = PHERO_INITIAL;
      phero[to + from * NODES] = PHERO_INITIAL;
    }
  } // end while that traverse all the lines in the file
}

__global__ void setupCurandStates(hiprandState *state_d, unsigned long t) {
  int gid = blockDim.x * blockIdx.x + threadIdx.x;
  hiprand_init(t, gid, 0, &state_d[gid]);
}

__global__ void initializeAnts(struct ant *antColony_d, hiprandState *state_d, float *bestSol_d) {

  int ant_id = blockDim.x * blockIdx.x + threadIdx.x;
  for (int node = 0; node < NODES; node++) {

    antColony_d[ant_id].tabu[node] =
        0; // set all nodes to nonvisited (0 means not in tabu list)
    antColony_d[ant_id].solution[node] =
        -1; // set all solution nodes as not in the solution (-1 means not in
            // solution)
  }
  bestSol_d[ant_id] = (float)TOTAL_DIST;
  // Select a the initial node randomly
  antColony_d[ant_id].curNode = hiprand(&state_d[ant_id]) % NODES;
  // Put the selected node in the solution list and in the tabu list
  antColony_d[ant_id].solution[0] = antColony_d[ant_id].curNode;
  antColony_d[ant_id].tabu[antColony_d[ant_id].curNode] =
      1; // 1 means that the node has been already visited
  antColony_d[ant_id].nextNode = -1; // we do not have a next node yet
  antColony_d[ant_id].solutionLen = 0;
  antColony_d[ant_id].pathIndex = 1;
}

__global__ void restartAnts(struct ant *antColony_d, hiprandState *state_d,
                            float *bestSol_d ) {

  int ant_id = blockDim.x * blockIdx.x + threadIdx.x;

  for (int node = 0; node < NODES; node++) {
    antColony_d[ant_id].tabu[node] =
        0; // set all nodes to nonvisited (0 means not in tabu list)
    antColony_d[ant_id].solution[node] =
        -1; // set all solution nodes as not in the solution (-1 means not in
            // solution)
  }
  if (antColony_d[ant_id].solutionLen < bestSol_d[ant_id] &&
      antColony_d[ant_id].solutionLen > 0) {
    bestSol_d[ant_id] = antColony_d[ant_id].solutionLen;

  }
  // Select a the initial node randomly
  antColony_d[ant_id].curNode = hiprand(&state_d[ant_id]) % NODES;
  // Put the selected node in the solution list and in the tabu list
  antColony_d[ant_id].solution[0] = antColony_d[ant_id].curNode;
  antColony_d[ant_id].tabu[antColony_d[ant_id].curNode] =
      1; // 1 means that the node has been already visited
  antColony_d[ant_id].nextNode = -1; // we do not have a next node yet
  antColony_d[ant_id].solutionLen = 0;
  antColony_d[ant_id].pathIndex = 1;
}

void acoSolve() {
  // This should iterate until the MAX_ITERATIONS number
  int iteration = 0;
  while (iteration++ < MAX_ITERATIONS) {
    // Part I (Solution construction phase)
    constructSolution<<<BLOCKS, THREADS>>>(antColony_d, state_d, heuristic_d,
                                           phero_d);

    hipDeviceSynchronize();
    // Move solution back to Host
    hipMemcpy(antColony, antColony_d, sizeof(antColony),
               hipMemcpyDeviceToHost);
    
   
    for (int i = 0; i < ANTS; i++) {
    }
    // Part II (Pheromone update process)
    // a. pheromone evaporation
    for (int from = 0; from < NODES; from++) {
      for (int to = 0; to < NODES; to++) {
        // only take the nodes that are different (if a node goes from 1 to 1
        // the len is 0 and we do not take care about this case)
        if (from != to) {
          phero[from + to * NODES] *= (1.0 - RHO);
          // if phero reach a negative value we restart it with the initial
          // value
          if (phero[from + to * NODES] < 0.0) {
            phero[from + to *NODES] = PHERO_INITIAL;
          }
        }
      } // end to for
    }   // end from for

    hipMemcpy(phero_d, phero, phero_size, hipMemcpyHostToDevice);
    hipMemcpy(bestSol, bestSol_d, sizeof(bestSol), hipMemcpyDeviceToHost);
    atomicUpdate<<<BLOCKS, THREADS>>>(antColony_d, phero_d);

    // traverse all the ants and get
    for (int i = 0; i < ANTS; i++) {
      if (bestSol[i] < globalBest) {
        globalBest = bestSol[i];
      }
    }

    restartAnts<<<BLOCKS, THREADS>>>(antColony_d, state_d, bestSol_d);
    hipDeviceSynchronize();

  } // end while iterations

  printf("Best Solution %f ", globalBest);
}

__global__ void atomicUpdate(struct ant *antColony_d, double *phero_d) {

  int ant_id = blockDim.x * blockIdx.x + threadIdx.x;
  int from, to;
  for (int i = 0; i < NODES; i++) {
    from = antColony_d[ant_id].solution[i];
    if (i > NODES - 1) {
      to = antColony_d[ant_id].solution[i + 1];
    } else {
      to = antColony_d[ant_id].solution[0];
    }
     atomicAdd(&phero_d[from + to * NODES], Q / antColony_d[ant_id].solutionLen * RHO);
     atomicAdd(&phero_d[from + to * NODES], Q / antColony_d[ant_id].solutionLen * RHO);
  }
}
__global__ void constructSolution(struct ant *antColony_d, hiprandState *state_d,
                                  float *heuristic_d, double *phero_d) {

  int ant_id = blockDim.x * blockIdx.x + threadIdx.x;
  int node = 0;

  while (node++ < NODES) {
    // Here we check if the solution is not complete (when the path Index is
    // equal to the number of nodes we are done)
    if (antColony_d[ant_id].pathIndex < NODES) {
      // Select the next node
      antColony_d[ant_id].nextNode =
          NextNode(antColony_d, ant_id, heuristic_d, phero_d, state_d);
      // Put the node in the tabu list and in the solution list
      antColony_d[ant_id].tabu[antColony_d[ant_id].nextNode] = 1;
      antColony_d[ant_id].solution[antColony_d[ant_id].pathIndex++] =
          antColony_d[ant_id].nextNode;
      // Add the distance to the solution Length
      antColony_d[ant_id].solutionLen +=
          heuristic_d[antColony_d[ant_id].curNode +
                      (antColony_d[ant_id].nextNode * NODES)];

      // In the case we get the last Node we get the distance from these last
      // node to the first node to gelin105t a closed tour
      if (antColony_d[ant_id].pathIndex == NODES) {
        antColony_d[ant_id].solutionLen +=
            heuristic_d[antColony_d[ant_id].solution[NODES - 1] +
                        (antColony_d[ant_id].solution[0] * NODES)];
      }
      // Now the new selected node is the current Node
      antColony_d[ant_id].curNode = antColony_d[ant_id].nextNode;
    }
  }
  // printf("ant len %f", antColony_d[2].solutionLen);
}

__device__ double probFunctionProduct(int from, int to, double *phero_d,
                                      float *heuristic_d) {
  double result;
  result = pow(phero_d[from + to * NODES], ALPHA) *
           pow(1 / (heuristic_d[from + to * NODES]), BETA);
  if (!isnan(result)) {
    return (double)((result));
  } else {
    return 0;
  }
}

__device__ int NextNode(struct ant *antColony_d, int pos, float *heuristic_d,
                        double *phero_d, hiprandState *state_d) {
  int to, from;
  double denom = 0.00000001;
  from = antColony_d[pos].curNode;
  for (to = 0; to < NODES; to++) {
    if (antColony_d[pos].tabu[to] == 0) {
      denom += probFunctionProduct(from, to, phero_d, heuristic_d);
    }
  }
  assert(denom != 0.0);
  to++;
  int count = NODES - antColony_d[pos].pathIndex;
  do {
    double p;
    to++;
    if (to >= NODES)
      to = 0;
    if (antColony_d[pos].tabu[to] ==
        0) { // 0 means not in tabu list (i.e., node enabled to participate in
             // selection)
      p = probFunctionProduct(from, to, phero_d, heuristic_d) / denom;
      double x = (double)(hiprand(&state_d[pos]) % 1000000000) / 1000000000.0;
      // When we get the roulette wheel selected element - break
      if (x < p) {
        break;
      }
      count--;
      if (count == 0) {
        break;
      }
    }
  } while (1);
  return to;
}

int main() {
  // The next section will handle the execution time record
  float exec_time;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Set blocks and threads based on number of ants
  if (ANTS <= 1024) {
    BLOCKS = 1;
    THREADS = ANTS;
  } else {
    THREADS = 1024;
    BLOCKS = ceil(ANTS / (float)THREADS);
  }
 //allocate host memory
  heuristic = (float *)malloc(NODES*NODES*sizeof(float));
  phero = (double*)malloc(NODES*NODES*sizeof(double));

  nodeTSP nodes[NODES];
  constructTSP("lin105", nodes);

 
  // allocate device memory
  hipMalloc((void **)&antColony_d, sizeof(antColony));
  hipMalloc((void **)&state_d, sizeof(state));
  hipMalloc((void **)&bestSol_d, sizeof(bestSol));
  hipMalloc((void **)&heuristic_d, heuristic_size);
  hipMalloc((void **)&phero_d, phero_size);

  hipMemcpy(heuristic_d, heuristic, heuristic_size, hipMemcpyHostToDevice);
  hipMemcpy(phero_d, phero, phero_size, hipMemcpyHostToDevice);

  // set curand states
  time_t t;
  time(&t);
  setupCurandStates<<<BLOCKS, THREADS>>>(state_d, (unsigned long)t);
  hipDeviceSynchronize();
  // Initialization phase
  initializeAnts<<<BLOCKS, THREADS>>>(antColony_d, state_d, bestSol_d);
  hipDeviceSynchronize();

  hipEventRecord(start, 0);
  // Construction phase
  acoSolve();

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&exec_time, start, stop);

  printf("%5.5f \n", exec_time / 1000); // time in ms is converted to seconds

  // Free memory
  free(phero);
  free(heuristic);

  hipFree(antColony_d);
  hipFree(heuristic_d);
  hipFree(phero_d);
  hipFree(state_d);
  hipFree(bestSol_d);

  return 0;
}
